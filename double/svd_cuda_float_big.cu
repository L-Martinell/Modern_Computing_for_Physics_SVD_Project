#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <string.h>
#include <hipsolver.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "matrix_utility.h"

#define CUDA_CHECK(call) \
  do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
      printf("CUDA error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
      exit(EXIT_FAILURE); \
    } \
  } while(0)


#define CUDA_CHECK_MSG(call, msg) \
  do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
      printf("CUDA error at %s:%d: %s | %s\n", __FILE__, __LINE__, hipGetErrorString(err), msg); \
      exit(EXIT_FAILURE); \
    } \
  } while (0)

void checkCuda(hipError_t result){
    if(result != hipSuccess){
        printf("Cuda runtime error: %s\n", hipGetErrorString(result));
        exit(-1);
    }
}

void checkCuSolver(hipsolverStatus_t status){
    if(status != HIPSOLVER_STATUS_SUCCESS){
        printf("cuSolver error\n");
        exit(-1);
    }
}


__global__ void MatrixTransposeKernel(int m, int n, float *A, float *T)
{
    // CUDA kernel to transpose a matrix. Useful bc cusolverDn expects a column-major indexing
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < m && col < n)
    {
        T[col * m + row] = A[row * n + col];
    }
}

__global__ void BuildSigmaKernel(int m, int n, int num_SV, float *lambda_list, float *Sigma, bool Eigenvalues){
    // Builds the matrix Sigma starting from the eigenvalues of A^T A or the singular values of A
    // NOTE -- This kernel works best with a 1D block, since it accesses data from a vector
    // Additionally, it requires that the matrix Sigma that is passed to it is initialized to 0's
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < num_SV){
        if(Eigenvalues){
            Sigma[index * n + index] = sqrtf(lambda_list[index]);
        }
        else{
            Sigma[index * n + index] = lambda_list[index];
        }
    }
}

__global__ void MatrixMultiplicationKernel(int m, int n, int k, float *A, float *B, float *C){
    // Multiplies two matrices A and B, not necessarily square
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    float sum = 0.0f;

    if(row < m && col < n){
        for(int l = 0; l < k; l++){
            sum += A[row * k + l] * B[l * n + col];
        }
    }
    C[row * n + col] = sum;
}


__global__ void MatrixSubtractionSquaredKernel(int m, int n, float *A, float *B, float *C){
    // This kernel calculates the square of the elementwise difference of two matrices A and B
    // C = (A - B)^2

    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    if(row < m && col < n){
        float diff = A[row * n + col] - B[row * n + col];
        C[row * n + col] = diff * diff;
    }
}

__global__ void MatrixReductionKernel(int m, int n, float *A, float *R){
    // This kernel performs the reduction of the matrix A

    extern __shared__ float partial_sum[];

    int tidx  = threadIdx.x;
    int bidx  = blockIdx.x;
    int bdim  = blockDim.x;
    int index = 2 * bdim * bidx;

    if(index >= m * n) return;

    partial_sum[tidx] = A[index + tidx];
    partial_sum[tidx + bdim] = A[index + tidx + bdim];

    for(int stride = bdim; stride > 0; stride /= 2){
        __syncthreads();
        if(tidx < stride){
            partial_sum[tidx] += partial_sum[tidx + stride];
        }
    }
    __syncthreads();
    if(tidx == 0){
        R[bidx] = partial_sum[0];
    }
}

int main()
{
    srand(13);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int m = 4096, n = 2048;
    int num_SV = (m < n) ? m : n;
    float *A     = (float*)malloc(sizeof(float) * m * n);

    generate_matrix(m, n, A);

    // Find the singular values of A, as well as the left and right matrices U and VT
    // Define the vector to keep the singular values, as well as the matrices U and VT
    float *s  = (float*)malloc(sizeof(float) * n);

    // Define the device quantities
    float *dA, *ds, *dU, *dVT;                                          // Row-major ordered matrices (useful for following calculations)
    // float *dA_cm;
    float *dU_cm, *dVT_cm;                                              // Column-major ordered matrices
    float *dA_rec;
    float *dSigma;                                                      // Singular value matrix
    float *dUSigma;

    // Allocate memory in the device (GPU)
    checkCuda(hipMalloc((void **)&dA, sizeof(float) * m * n));
    checkCuda(hipMalloc((void **)&ds, sizeof(float) * n));
    checkCuda(hipMalloc((void **)&dU, sizeof(float) * m * m));
    checkCuda(hipMalloc((void **)&dVT, sizeof(float)* n * n));
    checkCuda(hipMalloc((void **)&dA_rec, sizeof(float) * m * n));
    checkCuda(hipMalloc((void **)&dU_cm, sizeof(float) * m * m));
    checkCuda(hipMalloc((void **)&dVT_cm, sizeof(float)* n * n));
    checkCuda(hipMalloc((void **)&dSigma, sizeof(float) * m * n));
    checkCuda(hipMalloc((void **)&dUSigma, sizeof(float) * m * n));

    // Transfer the data from the host to the device
    checkCuda(hipMemcpy(dA, A, sizeof(float) * m * n, hipMemcpyHostToDevice));

    // Define the cuSolver object and create it
    hipsolverHandle_t solver = NULL;
    checkCuSolver(hipsolverDnCreate(&solver));

    // Define the worker quantities in the device
    int lwork = 0;
    hipsolverDnSgesvd_bufferSize(solver, m, n, &lwork);
    float *dWork;
    hipMalloc((void **)&dWork, sizeof(float) * lwork);

    int *dInfo;
    hipMalloc((void **)&dInfo, sizeof(int));

    // ######################################################## CUDA HEAVY STUFF ########################################################
    // Now transpose the matrix A
    float *dAT;
    hipMalloc((void **)&dAT, sizeof(float) * n * m);

    dim3 blockSize(16, 16);
    dim3 gridSize(n / blockSize.x, m / blockSize.y);

    // Start measuring time from here!
    hipEventRecord(start);

    MatrixTransposeKernel<<<gridSize, blockSize>>>(m, n, dA, dAT);

    // Now we can use dAT as the matrix to solve with cuSolver
    hipsolverStatus_t status = hipsolverDnSgesvd(
                                                solver,
                                                'A', 'A',
                                                m, n,
                                                dAT, m,
                                                ds,
                                                dU_cm, m,
                                                dVT_cm, n,
                                                dWork, lwork,
                                                NULL,
                                                dInfo);
    int info_gpu = 0;
    hipMemcpy(&info_gpu, dInfo, sizeof(int), hipMemcpyDeviceToHost);
    if(info_gpu != 0){
        printf("SVD failed, info = %d\n", info_gpu);
        exit(-1);
    }

    if (status != HIPSOLVER_STATUS_SUCCESS) {
        printf("hipsolverDnSgesvd failed with status %d\n", status);
    }

    // ############################ RECONSTRUCT MATRIX A ############################
    // A_rec = U * Sigma * VT

    // ############## BUILD MATRIX SIGMA ##############
    // Initialize dSigma to have all 0s
    int sigmaGridSize = 8;
    int sigmaThreadsPerBlock = num_SV / sigmaGridSize;

    checkCuda(hipMemset(dSigma, 0.0f, sizeof(float) * m * n));
    BuildSigmaKernel<<<sigmaGridSize, sigmaThreadsPerBlock>>>(m, n, num_SV, ds, dSigma, false);
    CUDA_CHECK_MSG(hipGetLastError(), "after BuildSigmaKernel");
    CUDA_CHECK(hipDeviceSynchronize());

    // ############## FROBENIUS NORM ##############
    // ####### TRANSPOSE MATRICES #######
    dim3 gridSize_mm((m + blockSize.x - 1) / blockSize.x, (m + blockSize.y - 1) / blockSize.y);
    dim3 gridSize_nn((n + blockSize.x - 1) / blockSize.x, (n + blockSize.y - 1) / blockSize.y);

    MatrixTransposeKernel<<<gridSize_mm, blockSize>>>(m, m, dU_cm, dU);
    MatrixTransposeKernel<<<gridSize_nn, blockSize>>>(n, n, dVT_cm, dVT);
    
    // ####### MULTIPLY MATRICES #######
    // U * Sigma = USigma first, then USigma * VT = A_rec
    dim3 gridSize2((n + blockSize.x - 1) / blockSize.x, (m + blockSize.y - 1) / blockSize.y);
    MatrixMultiplicationKernel<<<gridSize2, blockSize>>>(m, n, m, dU, dSigma, dUSigma);
    CUDA_CHECK_MSG(hipGetLastError(), "after MatrixMultiplicationKernel (U * Sigma)");
    CUDA_CHECK(hipDeviceSynchronize());

    MatrixMultiplicationKernel<<<gridSize2, blockSize>>>(m, n, n, dUSigma, dVT, dA_rec);
    CUDA_CHECK_MSG(hipGetLastError(), "after MatrixMultiplicationKernel (USigma * VT)");
    CUDA_CHECK(hipDeviceSynchronize());

    // ####### SUBTRACTION SQUARED #######
    float *Red = (float *)malloc(sizeof(float) * n);
    float *dDiff, *dRed;
    hipMalloc((void**)&dDiff, sizeof(float) * m * n);
    hipMalloc((void**)&dRed, sizeof(float) * n);

    MatrixSubtractionSquaredKernel<<<gridSize, blockSize>>>(m, n, dA, dA_rec, dDiff);
    CUDA_CHECK_MSG(hipGetLastError(), "after MatrixSubtractionSquaredKernel");
    CUDA_CHECK(hipDeviceSynchronize());

    // ####### REDUCTION #######
    int reductionThreadsPerBlock = m / 4, reductionGridSize = 4 * n;                // Make sure to cover the whole matrix while also not exceeding limitations
    MatrixReductionKernel<<<reductionGridSize, reductionThreadsPerBlock, 2 * reductionThreadsPerBlock * sizeof(float)>>>(m, n, dDiff, dRed);
    CUDA_CHECK_MSG(hipGetLastError(), "after MatrixReductionKernel");
    CUDA_CHECK(hipDeviceSynchronize());

    // Stop recording time here
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Fetch the data back to the host
    checkCuda(hipMemcpy(s, ds, sizeof(float) * n, hipMemcpyDeviceToHost));
    checkCuda(hipMemcpy(Red, dRed, sizeof(float) * n, hipMemcpyDeviceToHost));

    float FNorm = 0.0f;
    for(int i = 0; i < n; i++){
        FNorm += Red[i];
    }
    FNorm = sqrtf(FNorm);
    printf("Frobenius norm: %f\n", FNorm);

    float elapsed_time = 0.0;
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Elapsed time: %f s\n", elapsed_time/1000.0);
    printf("Matrix size: %d x %d\n", m, n);

    /*
    FILE *benchmark = fopen("benchmarking_float.txt", "a");
    fprintf(benchmark, "Matrix size: %d x %d, Frobenius norm: %f, time elapsed (SVD GPU float): %.10lf s ", m, n, FNorm, elapsed_time/1000.0);
    fprintf(benchmark, "MatrixTranspositionKernel (matrix A): gridSize = (%d, %d), blockSize = (%d, %d); ", gridSize.x, gridSize.y, blockSize.x, blockSize.y);
    fprintf(benchmark, "MatrixTranspositionKernel (matrix U): gridSize = (%d, %d), blockSize = (%d, %d); ", gridSize_mm.x, gridSize_mm.y, blockSize.x, blockSize.y);
    fprintf(benchmark, "MatrixTranspositionKernel (matrix V): gridSize = (%d, %d), blockSize = (%d, %d); ", gridSize_nn.x, gridSize_nn.y, blockSize.x, blockSize.y);
    fprintf(benchmark, "MatrixMultiplicationKernel: gridSize = (%d, %d), blockSize = (%d, %d); ", gridSize2.x, gridSize2.y, blockSize.x, blockSize.y);
    fprintf(benchmark, "MatrixReductionKernel: gridSize = %d, blockSize = %d, shared Memory size = %ld", reductionGridSize, reductionThreadsPerBlock, 2 * reductionThreadsPerBlock * sizeof(float));
    fclose(benchmark);
    */

    // Free the memory
    free(A);
    free(s);
    free(Red);

    hipFree(dA);
    hipFree(ds);
    hipFree(dU);
    hipFree(dVT);
    hipFree(dWork);
    hipFree(dInfo);
    hipFree(dSigma);
    hipFree(dA_rec);
    hipFree(dU_cm);
    hipFree(dVT_cm);
    hipFree(dUSigma);
    hipFree(dDiff);
    hipFree(dRed);
    hipFree(dAT);

    hipsolverDnDestroy(solver);

    return 0;
}